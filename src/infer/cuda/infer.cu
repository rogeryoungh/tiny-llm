#include "hip/hip_runtime.h"
#include "block_reduce.cuh"
#include "infer.hpp"

#include <cfloat>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                                                               \
  do {                                                                                                                 \
    hipError_t err = call;                                                                                            \
    if (err != hipSuccess) {                                                                                          \
      std::fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);                  \
      std::exit(-1);                                                                                                   \
    }                                                                                                                  \
  } while (0)

namespace tinyllm::cuda {

void *cuda_malloc(std::size_t size) {
  void *device = nullptr;
  CUDA_CHECK(hipMalloc(&device, size));
  return device;
}

void copy_to_device(const void *src, std::size_t size, void *dst) {
  CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void copy_to_host(const void *src, std::size_t size, void *dst) {
  CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

void cuda_free(void *ptr) { CUDA_CHECK(hipFree(ptr)); }

void *upload(const void *src, std::size_t size) {
  void *device = cuda_malloc(size);
  copy_to_device(src, size, device);
  return device;
}

void check_and_sync() {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
    std::exit(-1);
  }
  CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void rope_inplace_fp32_kernel(float *x, int head_dim, int pos, float theta) {
  const int half_dim = head_dim >> 1;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  int head = blockIdx.y;

  if (tid < half_dim) {
    float freq = powf(theta, -float(tid) / half_dim);
    float val = pos * freq;

    float vs, vc;
    sincosf(val, &vs, &vc);

    float *xh = x + head * head_dim;

    float v0 = xh[tid];
    float v1 = xh[tid + half_dim];

    xh[tid] = v0 * vc - v1 * vs;
    xh[tid + half_dim] = v0 * vs + v1 * vc;
  }
}

void rope_inplace_fp32(float *x, int num_heads, int head_dim, int pos, float theta) {
  const int half_dim = head_dim >> 1;

  const int threads_per_block = 256;

  int blocks_x = (half_dim + threads_per_block - 1) / threads_per_block;
  int blocks_y = int(num_heads);

  dim3 grid(blocks_x, blocks_y);
  dim3 block(threads_per_block);

  rope_inplace_fp32_kernel<<<grid, block>>>(x, head_dim, pos, theta);
}

__global__ void rms_norm_fp32_b_fp16_kernel(const float *x, half const *weight, float *out, int size, float eps) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  int batch = blockIdx.y;

  const float *xb = x + batch * size;
  float *outb = out + batch * size;

  float local_sum = 0.0f;
  for (int i = tid; i < size; i += stride) {
    float v = xb[i];
    local_sum += v * v;
  }
  local_sum = block_reduce_sum(local_sum);

  float inv_norm = rsqrtf(local_sum / float(size) + eps);

  for (int i = tid; i < size; i += stride) {
    float w = __half2float(weight[i]);
    outb[i] = xb[i] * inv_norm * w;
  }
}

void rms_norm_fp32_b_fp16(float *out, const float *x, const void *weight, int size, int num_batches, float eps) {
  int blocksize = std::max(std::min(1024, bit_ceil(size)), 32);

  dim3 grid(1, num_batches);
  dim3 block(blocksize);

  rms_norm_fp32_b_fp16_kernel<<<grid, block>>>(x, reinterpret_cast<const half *>(weight), out, size, eps);
}

__global__ void vec_add_inplace_fp32_kernel(float *a, const float *b, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) {
    a[tid] += b[tid];
  }
}

void vec_add_inplace_fp32(float *out, const float *a, int n) {
  const int block = 256;

  int grid = (n + block - 1) / block;

  vec_add_inplace_fp32_kernel<<<grid, block>>>(out, a, n);
}

__global__ void copy_fp16_to_fp32_n_kernel(const half *first, int n, float *result) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) {
    result[tid] = __half2float(first[tid]);
  }
}

__global__ void copy_fp32_to_fp16_n_kernel(const float *first, int n, half *result) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) {
    result[tid] = __float2half(first[tid]);
  }
}

void copy_fp16_to_fp32_n(const void *first, int n, float *result) {
  const int block = 256;
  int grid = (n + block - 1) / block;
  copy_fp16_to_fp32_n_kernel<<<grid, block>>>(reinterpret_cast<const half *>(first), n, result);
}

void copy_fp32_to_fp16_n(const float *first, int n, void *result) {
  const int block = 256;
  int grid = (n + block - 1) / block;
  copy_fp32_to_fp16_n_kernel<<<grid, block>>>(first, n, reinterpret_cast<half *>(result));
}

__global__ void swiglu_fp32_kernel(float *out, const float *x, const float *gate, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < size) {
    float g = gate[tid];
    float sig = 1.0f / (1.0f + expf(-g));
    float swish = g * sig;
    out[tid] = x[tid] * swish;
  }
}

void swiglu_fp32(float *out, const float *x, const float *gate, int size) {
  constexpr int TPB = 256;
  int n_blocks = (size + TPB - 1) / TPB;

  swiglu_fp32_kernel<<<n_blocks, TPB>>>(out, x, gate, size);
}
__global__ void compute_raw_scores(float *atth, const float *qh, const half *kh, int head_dim, int n_kv_heads,
                                   int kv_len) {
  const int col = blockIdx.x;
  if (col >= kv_len)
    return;

  const int lane = threadIdx.x;
  int kv_stride = n_kv_heads * head_dim;
  const half *kh_row = kh + col * kv_stride;
  float sum = 0.0f;
  float scale = rsqrtf(float(head_dim));

  for (int j = lane; j < head_dim; j += warpSize) {
    sum += qh[j] * __half2float(kh_row[j]);
  }

  sum = warp_reduce_sum(sum);

  if (lane == 0) {
    atth[col] = sum * scale;
  }
}

__global__ void softmax_inplace(float *atth, int kv_len) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  float max = -FLT_MAX;
  for (int i = tid; i < kv_len; i += stride)
    max = fmaxf(max, atth[i]);

  max = block_reduce_max(max);

  float sum = 0.0f;
  for (int i = tid; i < kv_len; i += stride) {
    float e = expf(atth[i] - max);
    atth[i] = e;
    sum += e;
  }
  sum = block_reduce_sum(sum);

  for (int i = tid; i < kv_len; i += stride)
    atth[i] /= sum;
}

__global__ void compute_weighted_sum(float *xout, const float *atth, const half *vh, int head_dim, int n_kv_heads,
                                     int kv_len) {
  const int col = blockIdx.x;
  if (col >= head_dim)
    return;

  const int lane = threadIdx.x;
  int kv_stride = n_kv_heads * head_dim;
  const half *vh_col = vh + col;
  float sum = 0.0f;

  for (int i = lane; i < kv_len; i += warpSize) {
    sum += atth[i] * __half2float(vh_col[i * kv_stride]);
  }

  sum = warp_reduce_sum(sum);

  if (lane == 0) {
    xout[col] = sum;
  }
}

template <int THREADS_PER_BLOCK>
__global__ void mh_compute_raw_scores(float *att, const float *q, const half *k, int head_dim, int n_kv_heads,
                                      int kv_len) {
  const int lane = threadIdx.x % warpSize;
  const int wid = threadIdx.x / warpSize;
  const int col = blockIdx.x * THREADS_PER_BLOCK + wid;
  if (col >= kv_len)
    return;

  const int head = blockIdx.y, n_heads = gridDim.y;
  const int q_per_head = n_heads / n_kv_heads;
  const float *qh = q + head * head_dim;
  float *atth = att + head * kv_len;
  const half *kh = k + (head / q_per_head) * head_dim;

  int kv_stride = n_kv_heads * head_dim;
  const half *kh_row = kh + col * kv_stride;
  float sum = 0.0f;
  float scale = rsqrtf(float(head_dim));

  for (int j = lane; j < head_dim; j += warpSize) {
    sum += qh[j] * __half2float(kh_row[j]);
  }

  sum = warp_reduce_sum(sum);

  if (lane == 0) {
    atth[col] = sum * scale;
  }
}

__global__ void mh_softmax_inplace(float *att, int kv_len) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  const int lane = threadIdx.x % warpSize;
  const int wid = threadIdx.x / warpSize;

  __shared__ float shared_max[32];
  __shared__ float shared_sum[32];

  int head = blockIdx.y;
  float *atth = att + head * kv_len;

  float max_val = -FLT_MAX;
  float sum_exp = 0.0f;
  for (int i = tid; i < kv_len; i += stride) {
    float ai = atth[i];
    float max = fmaxf(max_val, ai);
    sum_exp = sum_exp * expf(max_val - max) + expf(ai - max);
    max_val = max;
  }

  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    float max_other = __shfl_down_sync(0xFFFFFFFF, max_val, offset);
    float sum_other = __shfl_down_sync(0xFFFFFFFF, sum_exp, offset);

    float max = fmaxf(max_val, max_other);
    sum_exp = sum_exp * expf(max_val - max) + sum_other * expf(max_other - max);
    max_val = max;
  }

  if (lane == 0) {
    shared_max[wid] = max_val;
    shared_sum[wid] = sum_exp;
  }

  __syncthreads();

  if (wid == 0) {
    int warp_count = (blockDim.x + warpSize - 1) / warpSize;
    max_val = (threadIdx.x < warp_count) ? shared_max[lane] : -FLT_MAX;
    sum_exp = (threadIdx.x < warp_count) ? shared_sum[lane] : 0.0f;

    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
      float max_other = __shfl_down_sync(0xFFFFFFFF, max_val, offset);
      float sum_other = __shfl_down_sync(0xFFFFFFFF, sum_exp, offset);
      float max = fmaxf(max_val, max_other);
      sum_exp = sum_exp * expf(max_val - max) + sum_other * expf(max_other - max);
      max_val = max;
    }

    if (lane == 0) {
      shared_max[0] = max_val;
      shared_sum[0] = sum_exp;
    }
  }
  __syncthreads();

  max_val = shared_max[0];
  sum_exp = shared_sum[0];

  for (int i = tid; i < kv_len; i += stride) {
    float ai = atth[i];
    atth[i] = expf(ai - max_val) / sum_exp;
  }
}

template <int THREADS_PER_BLOCK>
__global__ void mh_compute_weighted_sum(float *out, const float *attn, const half *v, int head_dim, int n_kv_heads,
                                        int kv_len) {
  const int lane = threadIdx.x % warpSize;
  const int wid = threadIdx.x / warpSize;
  const int col = blockIdx.x * THREADS_PER_BLOCK + wid;
  if (col >= head_dim)
    return;

  const int head = blockIdx.y, n_heads = gridDim.y;
  const int q_per_head = n_heads / n_kv_heads;
  const float *atth = attn + head * kv_len;
  float *xout = out + head * head_dim;

  const half *vh = v + (head / q_per_head) * head_dim;

  int kv_stride = n_kv_heads * head_dim;
  const half *vh_col = vh + col;
  float sum = 0.0f;

  for (int i = lane; i < kv_len; i += warpSize) {
    sum += atth[i] * __half2float(vh_col[i * kv_stride]);
  }

  sum = warp_reduce_sum(sum);

  if (lane == 0) {
    xout[col] = sum;
  }
}

void attention_softmax_fp32_kv_fp16(float *out, float *atth, const float *qh, const void *kh, const void *vh,
                                    int head_dim, int n_kv_heads, int kv_len) {
  constexpr int WARP_SIZE = 32;

  compute_raw_scores<<<kv_len, WARP_SIZE>>>(atth, qh, reinterpret_cast<const half *>(kh), head_dim, n_kv_heads, kv_len);

  int block = std::max(32, std::min(bit_ceil(kv_len), 256));

  softmax_inplace<<<1, block>>>(atth, kv_len);

  compute_weighted_sum<<<head_dim, WARP_SIZE>>>(out, atth, reinterpret_cast<const half *>(vh), head_dim, n_kv_heads,
                                                kv_len);
}

void mh_attention_fp32_kv_fp16(float *out, float *att, const float *q, const void *k, const void *v, int num_heads,
                               int head_dim, int n_kv_heads, int kv_len) {
  constexpr int WARP_SIZE = 32, THREADS_PER_BLOCK = 4;
  dim3 grid1((kv_len + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, num_heads);
  mh_compute_raw_scores<THREADS_PER_BLOCK><<<grid1, WARP_SIZE * THREADS_PER_BLOCK>>>(
      att, q, reinterpret_cast<const half *>(k), head_dim, n_kv_heads, kv_len);
  dim3 grid2(1, num_heads);
  mh_softmax_inplace<<<grid2, 256>>>(att, kv_len);

  dim3 grid3((head_dim + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, num_heads);
  mh_compute_weighted_sum<THREADS_PER_BLOCK><<<grid3, WARP_SIZE * THREADS_PER_BLOCK>>>(
      out, att, reinterpret_cast<const half *>(v), head_dim, n_kv_heads, kv_len);
}

} // namespace tinyllm::cuda
